#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2020, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/gmres_mixed_kernels.hpp"


#include <algorithm>


#include <ginkgo/core/base/exception_helpers.hpp>
#include <ginkgo/core/base/math.hpp>
#include <ginkgo/core/base/range_accessors.hpp>
#include <ginkgo/core/base/std_extensions.hpp>
#include <ginkgo/core/matrix/dense.hpp>


#include "core/components/fill_array.hpp"
#include "cuda/base/config.hpp"
#include "cuda/base/cublas_bindings.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"
#include "cuda/components/atomic.cuh"
#include "cuda/components/cooperative_groups.cuh"
#include "cuda/components/reduction.cuh"
#include "cuda/components/thread_ids.cuh"
#include "cuda/components/uninitialized_array.hpp"


#include <iostream>


// #define TIMING 1


#ifdef TIMING
using double_seconds = std::chrono::duration<double, std::milli>;
#endif


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The GMRES_MIXED solver namespace.
 *
 * @ingroup gmres_mixed
 */
namespace gmres_mixed {


constexpr int default_block_size = 512;
// default_dot_dim can not be 64 in hip because 64 * 64 exceeds their max block
// size limit.
constexpr int default_dot_dim = 32;
constexpr int default_dot_size = default_dot_dim * default_dot_dim;


constexpr int default_update_dim = 32;


#include "common/solver/gmres_mixed_kernels.hpp.inc"


// Specialization, so the Accessor can use the same function as regular pointers
template <typename Type1, typename Type2>
xstd::enable_if_t<!Accessor2d<Type1, Type2>::has_scalar,
                  Accessor2d<cuda_type<Type1>, cuda_type<Type2>>>
as_cuda_accessor(Accessor2d<Type1, Type2> acc)
{
    return {as_cuda_type(acc.get_storage()), acc.get_stride()};
}

template <typename Type1, typename Type2>
xstd::enable_if_t<Accessor2d<Type1, Type2>::has_scalar,
                  Accessor2d<cuda_type<Type1>, cuda_type<Type2>>>
as_cuda_accessor(Accessor2d<Type1, Type2> acc)
{
    return {as_cuda_type(acc.get_storage()), acc.get_stride(),
            as_cuda_type(acc.get_scale())};
}

template <typename Type1, typename Type2>
xstd::enable_if_t<!Accessor2dConst<Type1, Type2>::has_scalar,
                  Accessor2dConst<cuda_type<Type1>, cuda_type<Type2>>>
as_cuda_accessor(const Accessor2dConst<Type1, Type2> &acc)
{
    return {as_cuda_type(acc.get_storage()), acc.get_stride()};
}

template <typename Type1, typename Type2>
xstd::enable_if_t<Accessor2dConst<Type1, Type2>::has_scalar,
                  Accessor2dConst<cuda_type<Type1>, cuda_type<Type2>>>
as_cuda_accessor(const Accessor2dConst<Type1, Type2> &acc)
{
    return {as_cuda_type(acc.get_storage()), acc.get_stride(),
            as_cuda_type(acc.get_scale())};
}


template <typename ValueType>
void zero_matrix(size_type m, size_type n, size_type stride, ValueType *array)
{
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(ceildiv(n, block_size.x), 1, 1);
    zero_matrix_kernel<<<grid_size, block_size, 0, 0>>>(m, n, stride,
                                                        as_cuda_type(array));
}


template <typename ValueType>
void initialize_1(std::shared_ptr<const CudaExecutor> exec,
                  const matrix::Dense<ValueType> *b,
                  matrix::Dense<ValueType> *b_norm,
                  matrix::Dense<ValueType> *residual,
                  matrix::Dense<ValueType> *givens_sin,
                  matrix::Dense<ValueType> *givens_cos,
                  Array<stopping_status> *stop_status, size_type krylov_dim)
{
    const auto num_threads = std::max(b->get_size()[0] * b->get_stride(),
                                      krylov_dim * b->get_size()[1]);
    const dim3 grid_dim(ceildiv(num_threads, default_block_size), 1, 1);
    const dim3 block_dim(default_block_size, 1, 1);
    constexpr auto block_size = default_block_size;

    b->compute_norm2(b_norm);
    initialize_1_kernel<block_size><<<grid_dim, block_dim>>>(
        b->get_size()[0], b->get_size()[1], krylov_dim,
        as_cuda_type(b->get_const_values()), b->get_stride(),
        as_cuda_type(residual->get_values()), residual->get_stride(),
        as_cuda_type(givens_sin->get_values()), givens_sin->get_stride(),
        as_cuda_type(givens_cos->get_values()), givens_cos->get_stride(),
        as_cuda_type(stop_status->get_data()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(
    GKO_DECLARE_GMRES_MIXED_INITIALIZE_1_KERNEL);


template <typename ValueType, typename ValueTypeKrylovBases>
void initialize_2(std::shared_ptr<const CudaExecutor> exec,
                  const matrix::Dense<ValueType> *residual,
                  matrix::Dense<ValueType> *residual_norm,
                  matrix::Dense<ValueType> *residual_norm_collection,
                  Accessor2d<ValueTypeKrylovBases, ValueType> krylov_bases,
                  matrix::Dense<ValueType> *next_krylov_basis,
                  Array<size_type> *final_iter_nums, size_type krylov_dim)
{
    const auto num_rows = residual->get_size()[0];
    const auto num_rhs = residual->get_size()[1];
    const dim3 grid_dim_1(
        ceildiv(num_rows * krylov_bases.get_stride(), default_block_size), 1,
        1);
    const dim3 block_dim(default_block_size, 1, 1);
    constexpr auto block_size = default_block_size;

    initialize_2_1_kernel<block_size><<<grid_dim_1, block_dim>>>(
        residual->get_size()[0], residual->get_size()[1], krylov_dim,
        as_cuda_accessor(krylov_bases),
        as_cuda_type(residual_norm_collection->get_values()),
        residual_norm_collection->get_stride());
    residual->compute_norm2(residual_norm);

    const dim3 grid_dim_2(ceildiv(num_rows * num_rhs, default_block_size), 1,
                          1);
    initialize_2_2_kernel<block_size><<<grid_dim_2, block_dim>>>(
        residual->get_size()[0], residual->get_size()[1],
        as_cuda_type(residual->get_const_values()), residual->get_stride(),
        as_cuda_type(residual_norm->get_const_values()),
        as_cuda_type(residual_norm_collection->get_values()),
        as_cuda_accessor(krylov_bases),
        as_cuda_type(next_krylov_basis->get_values()),
        next_krylov_basis->get_stride(),
        as_cuda_type(final_iter_nums->get_data()));
}

GKO_INSTANTIATE_FOR_EACH_GMRES_MIXED_TYPE(
    GKO_DECLARE_GMRES_MIXED_INITIALIZE_2_KERNEL);


template <typename ValueType, typename ValueTypeKrylovBases>
void finish_arnoldi(std::shared_ptr<const CudaExecutor> exec,
                    matrix::Dense<ValueType> *next_krylov_basis,
                    Accessor2d<ValueTypeKrylovBases, ValueType> krylov_bases,
                    matrix::Dense<ValueType> *hessenberg_iter, size_type iter,
                    const stopping_status *stop_status)
{
    const auto stride_next_krylov = next_krylov_basis->get_stride();
    const auto stride_krylov = krylov_bases.get_stride();
    const auto stride_hessenberg = hessenberg_iter->get_stride();
    const auto dim_size = next_krylov_basis->get_size();
    auto cublas_handle = exec->get_cublas_handle();
    const dim3 grid_size(ceildiv(dim_size[1], default_dot_dim),
                         exec->get_num_multiprocessor() * 2);
    const dim3 block_size(default_dot_dim, default_dot_dim);
    for (size_type k = 0; k < iter + 1; ++k) {
        components::fill_array(
            exec, hessenberg_iter->get_values() + k * stride_hessenberg,
            dim_size[1], zero<ValueType>());
        multidot_kernel<<<grid_size, block_size>>>(
            k, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_const_values()),
            stride_next_krylov, as_cuda_accessor(krylov_bases),
            as_cuda_type(hessenberg_iter->get_values()), stride_hessenberg,
            as_cuda_type(stop_status));
        update_next_krylov_kernel<default_block_size>
            <<<ceildiv(dim_size[0] * stride_next_krylov, default_block_size),
               default_block_size>>>(
                k, dim_size[0], dim_size[1],
                as_cuda_type(next_krylov_basis->get_values()),
                stride_next_krylov, as_cuda_accessor(krylov_bases),
                as_cuda_type(hessenberg_iter->get_const_values()),
                stride_hessenberg, as_cuda_type(stop_status));
    }
    // for i in 1:iter
    //     hessenberg(iter, i) = next_krylov_basis' * krylov_bases(:, i)
    //     next_krylov_basis  -= hessenberg(iter, i) * krylov_bases(:, i)
    // end


    update_hessenberg_2_kernel<default_block_size>
        <<<dim_size[1], default_block_size>>>(
            iter, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_const_values()),
            stride_next_krylov, as_cuda_type(hessenberg_iter->get_values()),
            stride_hessenberg, as_cuda_type(stop_status));

    update_krylov_next_krylov_kernel<default_block_size>
        <<<ceildiv(dim_size[0] * stride_next_krylov, default_block_size),
           default_block_size>>>(
            iter, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_values()), stride_next_krylov,
            as_cuda_accessor(krylov_bases),
            as_cuda_type(hessenberg_iter->get_const_values()),
            stride_hessenberg, as_cuda_type(stop_status));
    // next_krylov_basis /= hessenberg(iter, iter + 1)
    // krylov_bases(:, iter + 1) = next_krylov_basis
    // End of arnoldi
}


template <typename ValueType, typename ValueTypeKrylovBases>
void finish_arnoldi_reorth(
    std::shared_ptr<const CudaExecutor> exec,
    matrix::Dense<ValueType> *next_krylov_basis,
    Accessor2d<ValueTypeKrylovBases, ValueType> krylov_bases,
    matrix::Dense<ValueType> *hessenberg_iter,
    matrix::Dense<ValueType> *buffer_iter,
    matrix::Dense<ValueType> *arnoldi_norm, size_type iter,
    const stopping_status *stop_status, stopping_status *reorth_status,
    Array<size_type> *num_reorth)
{
    const auto stride_next_krylov = next_krylov_basis->get_stride();
    const auto stride_krylov = krylov_bases.get_stride();
    const auto stride_hessenberg = hessenberg_iter->get_stride();
    const auto stride_buffer = buffer_iter->get_stride();
    const auto stride_arnoldi = arnoldi_norm->get_stride();
    const auto dim_size = next_krylov_basis->get_size();
    auto cublas_handle = exec->get_cublas_handle();
    const dim3 grid_size(ceildiv(dim_size[1], default_dot_dim),
                         exec->get_num_multiprocessor() * 2);
    const dim3 block_size(default_dot_dim, default_dot_dim);
    size_type numReorth;
    Accessor2d<ValueType, ValueType> next_krylov_accessor{
        next_krylov_basis->get_values(), stride_next_krylov};
    auto next_krylov_const_accessor = next_krylov_accessor.to_const();
    //    size_type num_reorth;
    components::fill_array(exec, arnoldi_norm->get_values(), dim_size[1],
                           zero<ValueType>());
    multidot_kernel<<<grid_size, block_size>>>(
        0, dim_size[0], dim_size[1],
        as_cuda_type(next_krylov_basis->get_const_values()), stride_next_krylov,
        as_cuda_accessor(next_krylov_const_accessor),
        as_cuda_type(next_krylov_basis->get_const_values()), stride_next_krylov,
        as_cuda_type(arnoldi_norm->get_values()), 0, as_cuda_type(stop_status));
    // nrm = norm(next_krylov_basis
    for (size_type k = 0; k < iter + 1; ++k) {
        components::fill_array(
            exec, hessenberg_iter->get_values() + k * stride_hessenberg,
            dim_size[1], zero<ValueType>());
        multidot_kernel<<<grid_size, block_size>>>(
            k, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_const_values()),
            stride_next_krylov, as_cuda_accessor(krylov_bases),
            as_cuda_type(hessenberg_iter->get_values()), stride_hessenberg,
            as_cuda_type(stop_status));
        components::fill_array(exec, num_reorth->get_data(), 1,
                               zero<size_type>());
        update_next_krylov_kernel_and_check<default_block_size>
            <<<ceildiv(dim_size[0] * stride_next_krylov, default_block_size),
               default_block_size>>>(
                k, dim_size[0], dim_size[1],
                as_cuda_type(next_krylov_basis->get_values()),
                stride_next_krylov, as_cuda_accessor(krylov_bases),
                as_cuda_type(hessenberg_iter->get_const_values()),
                stride_hessenberg, as_cuda_type(arnoldi_norm->get_values()), 0,
                as_cuda_type(stop_status), as_cuda_type(reorth_status),
                as_cuda_type(num_reorth->get_data()));
        numReorth = 0;
        exec->get_master()->copy_from(exec.get(), 1,
                                      num_reorth->get_const_data(), &numReorth);
        if (numReorth > 0) {
            components::fill_array(
                exec, buffer_iter->get_values() + k * stride_buffer,
                dim_size[1], zero<ValueType>());
            multidot_kernel<<<grid_size, block_size>>>(
                k, dim_size[0], dim_size[1],
                as_cuda_type(next_krylov_basis->get_const_values()),
                stride_next_krylov, as_cuda_accessor(krylov_bases),
                as_cuda_type(buffer_iter->get_values()), stride_buffer,
                as_cuda_type(stop_status));
            update_next_krylov_kernel_and_add<default_block_size><<<
                ceildiv(dim_size[0] * stride_next_krylov, default_block_size),
                default_block_size>>>(
                k, dim_size[0], dim_size[1],
                as_cuda_type(next_krylov_basis->get_values()),
                stride_next_krylov, as_cuda_accessor(krylov_bases),
                as_cuda_type(hessenberg_iter->get_values()), stride_hessenberg,
                as_cuda_type(buffer_iter->get_const_values()), stride_buffer,
                as_cuda_type(stop_status), as_cuda_type(reorth_status));
        }
    }
    // for i in 1:iter
    //     hessenberg(iter, i) = next_krylov_basis' * krylov_bases(:, i)
    //     next_krylov_basis  -= hessenberg(iter, i) * krylov_bases(:, i)
    //     if (hessenberg(iter, i)*hessenberg(iter, i)>nrm*0.99)
    //         buffer = next_krylov_basis' * krylov_bases(:, i)
    //         next_krylov_basis  -= buffer * krylov_bases(:, i)
    //         hessenberg(iter, i) += buffer;
    //     end
    // end


    update_hessenberg_2_kernel<default_block_size>
        <<<dim_size[1], default_block_size>>>(
            iter, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_const_values()),
            stride_next_krylov, as_cuda_type(hessenberg_iter->get_values()),
            stride_hessenberg, as_cuda_type(stop_status));

    update_krylov_next_krylov_kernel<default_block_size>
        <<<ceildiv(dim_size[0] * stride_next_krylov, default_block_size),
           default_block_size>>>(
            iter, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_values()), stride_next_krylov,
            as_cuda_accessor(krylov_bases),
            as_cuda_type(hessenberg_iter->get_const_values()),
            stride_hessenberg, as_cuda_type(stop_status));
    // next_krylov_basis /= hessenberg(iter, iter + 1)
    // krylov_bases(:, iter + 1) = next_krylov_basis
    // End of arnoldi
}


template <typename ValueType, typename ValueTypeKrylovBases>
void finish_arnoldi_CGS(
    std::shared_ptr<const CudaExecutor> exec,
    matrix::Dense<ValueType> *next_krylov_basis,
    Accessor2d<ValueTypeKrylovBases, ValueType> krylov_bases,
    matrix::Dense<ValueType> *hessenberg_iter,
    matrix::Dense<ValueType> *buffer_iter,
    matrix::Dense<ValueType> *arnoldi_norm, size_type iter,
    const stopping_status *stop_status, stopping_status *reorth_status,
    Array<size_type> *num_reorth)
{
    const auto stride_next_krylov = next_krylov_basis->get_stride();
    const auto stride_krylov = krylov_bases.get_stride();
    const auto stride_hessenberg = hessenberg_iter->get_stride();
    const auto stride_buffer = buffer_iter->get_stride();
    const auto stride_arnoldi = arnoldi_norm->get_stride();
    const auto dim_size = next_krylov_basis->get_size();
    auto cublas_handle = exec->get_cublas_handle();
    const dim3 grid_size(ceildiv(dim_size[1], default_dot_dim),
                         exec->get_num_multiprocessor() * 2);
    const dim3 block_size(default_dot_dim, default_dot_dim);
    size_type numReorth;
    Accessor2d<ValueType, ValueType> next_krylov_accessor{
        next_krylov_basis->get_values(), stride_next_krylov};
    auto next_krylov_const_accessor = next_krylov_accessor.to_const();

    components::fill_array(exec, arnoldi_norm->get_values(), dim_size[1],
                           zero<ValueType>());
    multidot_kernel<<<grid_size, block_size>>>(
        0, dim_size[0], dim_size[1],
        as_cuda_type(next_krylov_basis->get_const_values()), stride_next_krylov,
        as_cuda_accessor(next_krylov_const_accessor),
        as_cuda_type(arnoldi_norm->get_values()), 0, as_cuda_type(stop_status));
    // nrmP = norm(next_krylov_basis
    for (size_type k = 0; k < iter + 1; ++k) {
        components::fill_array(
            exec, hessenberg_iter->get_values() + k * stride_hessenberg,
            dim_size[1], zero<ValueType>());
        multidot_kernel<<<grid_size, block_size>>>(
            k, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_const_values()),
            stride_next_krylov, as_cuda_accessor(krylov_bases),
            as_cuda_type(hessenberg_iter->get_values()), stride_hessenberg,
            as_cuda_type(stop_status));
    }
    // for i in 1:iter
    //     hessenberg(iter, i) = next_krylov_basis' * krylov_bases(:, i)
    // end
    for (size_type k = 0; k < iter + 1; ++k) {
        update_next_krylov_kernel<default_block_size>
            <<<ceildiv(dim_size[0] * stride_next_krylov, default_block_size),
               default_block_size>>>(
                k, dim_size[0], dim_size[1],
                as_cuda_type(next_krylov_basis->get_values()),
                stride_next_krylov, as_cuda_accessor(krylov_bases),
                as_cuda_type(hessenberg_iter->get_const_values()),
                stride_hessenberg, as_cuda_type(stop_status));
    }
    // for i in 1:iter
    //     next_krylov_basis  -= hessenberg(iter, i) * krylov_bases(:, i)
    // end
    components::fill_array(exec, arnoldi_norm->get_values() + dim_size[1],
                           dim_size[1], zero<ValueType>());
    multidot_kernel<<<grid_size, block_size>>>(
        0, dim_size[0], dim_size[1],
        as_cuda_type(next_krylov_basis->get_const_values()), stride_next_krylov,
        as_cuda_accessor(next_krylov_const_accessor),
        as_cuda_type(arnoldi_norm->get_values() + dim_size[1]), 0,
        as_cuda_type(stop_status));
    components::fill_array(exec, num_reorth->get_data(), 1, zero<size_type>());
    check_arnoldi_norms<default_block_size>
        <<<ceildiv(dim_size[1], default_block_size), default_block_size>>>(
            as_cuda_type(arnoldi_norm->get_const_values()), stride_arnoldi,
            as_cuda_type(arnoldi_norm->get_const_values() + stride_arnoldi),
            stride_arnoldi, as_cuda_type(stop_status),
            as_cuda_type(reorth_status), as_cuda_type(num_reorth->get_data()));
    numReorth = 0;
    exec->get_master()->copy_from(exec.get(), 1, num_reorth->get_const_data(),
                                  &numReorth);
    // numReorth <= number of next_krylov vector to be reorthogonalization
    for (size_type l = 1; (numReorth > 0) && (l < 3); l++) {
        for (size_type k = 0; k < iter + 1; ++k) {
            components::fill_array(
                exec, buffer_iter->get_values() + k * stride_buffer,
                dim_size[1], zero<ValueType>());
            multidot_kernel<<<grid_size, block_size>>>(
                k, dim_size[0], dim_size[1],
                as_cuda_type(next_krylov_basis->get_const_values()),
                stride_next_krylov, as_cuda_accessor(krylov_bases),
                as_cuda_type(buffer_iter->get_values()), stride_buffer,
                as_cuda_type(stop_status));
        }
        // for i in 1:iter
        //     hessenberg(iter, i) = next_krylov_basis' * krylov_bases(:, i)
        // end
        for (size_type k = 0; k < iter + 1; ++k) {
            update_next_krylov_kernel_and_add<default_block_size><<<
                ceildiv(dim_size[0] * stride_next_krylov, default_block_size),
                default_block_size>>>(
                k, dim_size[0], dim_size[1],
                as_cuda_type(next_krylov_basis->get_values()),
                stride_next_krylov, as_cuda_accessor(krylov_bases),
                as_cuda_type(hessenberg_iter->get_values()), stride_hessenberg,
                as_cuda_type(buffer_iter->get_const_values()), stride_buffer,
                as_cuda_type(stop_status), as_cuda_type(reorth_status));
        }
        // for i in 1:iter
        //     next_krylov_basis  -= hessenberg(iter, i) * krylov_bases(:, i)
        // end
        components::fill_array(
            exec, arnoldi_norm->get_values() + dim_size[1] * (l - 1),
            dim_size[1], zero<ValueType>());
        multidot_kernel<<<grid_size, block_size>>>(
            0, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_const_values()),
            stride_next_krylov, as_cuda_accessor(next_krylov_const_accessor),
            as_cuda_type(arnoldi_norm->get_values() + dim_size[1] * (l - 1)), 0,
            as_cuda_type(stop_status));
        // nrmN = norm(next_krylov_basis)
        components::fill_array(exec, num_reorth->get_data(), 1,
                               zero<size_type>());
        check_arnoldi_norms<default_block_size>
            <<<ceildiv(dim_size[1], default_block_size), default_block_size>>>(
                as_cuda_type(arnoldi_norm->get_const_values() +
                             stride_arnoldi * (2 - l)),
                stride_arnoldi,
                as_cuda_type(arnoldi_norm->get_const_values() +
                             stride_arnoldi * (l - 1)),
                stride_arnoldi, as_cuda_type(stop_status),
                as_cuda_type(reorth_status),
                as_cuda_type(num_reorth->get_data()));
        numReorth = 0;
        exec->get_master()->copy_from(exec.get(), 1,
                                      num_reorth->get_const_data(), &numReorth);
        // numReorth <= number of next_krylov vector to be reorthogonalization
    }


    update_hessenberg_2_kernel<default_block_size>
        <<<dim_size[1], default_block_size>>>(
            iter, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_const_values()),
            stride_next_krylov, as_cuda_type(hessenberg_iter->get_values()),
            stride_hessenberg, as_cuda_type(stop_status));
    update_krylov_next_krylov_kernel<default_block_size>
        <<<ceildiv(dim_size[0] * stride_next_krylov, default_block_size),
           default_block_size>>>(
            iter, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_values()), stride_next_krylov,
            as_cuda_accessor(krylov_bases),
            as_cuda_type(hessenberg_iter->get_const_values()),
            stride_hessenberg, as_cuda_type(stop_status));
    // next_krylov_basis /= hessenberg(iter, iter + 1)
    // krylov_bases(:, iter + 1) = next_krylov_basis
    // End of arnoldi
}
/**/


template <typename ValueType, typename ValueTypeKrylovBases>
void finish_arnoldi_CGS2(
    std::shared_ptr<const CudaExecutor> exec,
    matrix::Dense<ValueType> *next_krylov_basis,
    Accessor2d<ValueTypeKrylovBases, ValueType> krylov_bases,
    matrix::Dense<ValueType> *hessenberg_iter,
    matrix::Dense<ValueType> *buffer_iter,
    matrix::Dense<ValueType> *arnoldi_norm, size_type iter,
    const stopping_status *stop_status, stopping_status *reorth_status,
    Array<size_type> *num_reorth, int *num_reorth_steps,
    int *num_reorth_vectors)
{
    const auto stride_next_krylov = next_krylov_basis->get_stride();
    const auto stride_krylov = krylov_bases.get_stride();
    const auto stride_hessenberg = hessenberg_iter->get_stride();
    const auto stride_buffer = buffer_iter->get_stride();
    const auto stride_arnoldi = arnoldi_norm->get_stride();
    const auto dim_size = next_krylov_basis->get_size();
    auto cublas_handle = exec->get_cublas_handle();
    const dim3 grid_size(ceildiv(dim_size[1], default_dot_dim),
                         exec->get_num_multiprocessor() * 2);
    const dim3 grid_size_num_iters(
        ceildiv(dim_size[1] * (iter + 1), default_dot_dim),
        exec->get_num_multiprocessor() * 2);
    const dim3 block_size(default_dot_dim, default_dot_dim);
    size_type numReorth;

    Accessor2d<ValueType, ValueType> next_krylov_accessor{
        next_krylov_basis->get_values(), stride_next_krylov};
    auto next_krylov_const_accessor = next_krylov_accessor.to_const();

    components::fill_array(exec, arnoldi_norm->get_values(), dim_size[1],
                           zero<ValueType>());
    multidot_kernel<<<grid_size, block_size>>>(
        0, dim_size[0], dim_size[1],
        as_cuda_type(next_krylov_basis->get_const_values()), stride_next_krylov,
        as_cuda_accessor(next_krylov_const_accessor),
        // as_cuda_type(next_krylov_basis->get_const_values()),
        // stride_next_krylov,
        as_cuda_type(arnoldi_norm->get_values()), 0, as_cuda_type(stop_status));
    // nrmP = norm(next_krylov_basis
#ifdef TIMING
    exec->synchronize();
    auto start_1 = std::chrono::steady_clock::now();
#endif
    /* */
    zero_matrix(iter + 1, dim_size[1], stride_hessenberg,
                hessenberg_iter->get_values());
    /*
    multidot_kernel_num_iters<<<grid_size, block_size>>>(
        iter + 1, dim_size[0], dim_size[1],
        as_cuda_type(next_krylov_basis->get_const_values()), stride_next_krylov,
        as_cuda_accessor(krylov_bases.to_const()),
        as_cuda_type(hessenberg_iter->get_values()), stride_hessenberg,
        as_cuda_type(stop_status));
    */
    /* */
    //    write(std::cout, krylov_bases);
    //    write(std::cout, next_krylov_basis);
    multidot_kernel_num_iters_1<<<grid_size_num_iters, block_size>>>(
        iter + 1, dim_size[0], dim_size[1],
        as_cuda_type(next_krylov_basis->get_const_values()), stride_next_krylov,
        as_cuda_accessor(krylov_bases.to_const()),
        as_cuda_type(hessenberg_iter->get_values()), stride_hessenberg,
        as_cuda_type(stop_status));
    // exec->synchronize();
    // write(std::cout, hessenberg_iter);
    /* */
#ifdef TIMING
    exec->synchronize();
    auto time_1 = std::chrono::steady_clock::now() - start_1;
    std::cout << "time_1(" << iter << ") = "
              << std::chrono::duration_cast<double_seconds>(time_1).count()
              << std::endl;
#endif
    // for i in 1:iter
    //     hessenberg(iter, i) = next_krylov_basis' * krylov_bases(:, i)
    // end
#ifdef TIMING
    exec->synchronize();
    auto start_2 = std::chrono::steady_clock::now();
#endif
    /*
    update_next_krylov_kernel_num_iters<default_block_size>
        <<<ceildiv(dim_size[0] * stride_next_krylov, default_block_size),
           default_block_size>>>(
            iter + 1, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_values()), stride_next_krylov,
            as_cuda_accessor(krylov_bases.to_const()),
            as_cuda_type(hessenberg_iter->get_const_values()),
            stride_hessenberg, as_cuda_type(stop_status));
    */
    /* */
    // update_next_krylov_kernel_num_iters_1<default_block_size>
    //     <<<dim_size[0], default_block_size>>>(
    update_next_krylov_kernel_num_iters_1<default_update_dim>
        <<<dim_size[0] * stride_next_krylov, default_update_dim>>>(
            iter + 1, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_values()), stride_next_krylov,
            as_cuda_accessor(krylov_bases.to_const()),
            as_cuda_type(hessenberg_iter->get_const_values()),
            stride_hessenberg, as_cuda_type(stop_status));
    /* */
#ifdef TIMING
    exec->synchronize();
    auto time_2 = std::chrono::steady_clock::now() - start_2;
    std::cout << "time_2(" << iter << ") = "
              << std::chrono::duration_cast<double_seconds>(time_2).count()
              << std::endl;
    std::cout << "time_1 / time_2(" << iter << ") = "
              << std::chrono::duration_cast<double_seconds>(time_1).count() /
                     std::chrono::duration_cast<double_seconds>(time_2).count()
              << std::endl;
#endif
    // for i in 1:iter
    //     next_krylov_basis  -= hessenberg(iter, i) * krylov_bases(:, i)
    // end
    components::fill_array(exec, arnoldi_norm->get_values() + dim_size[1],
                           dim_size[1], zero<ValueType>());
#ifdef TIMING
    exec->synchronize();
    auto start_3 = std::chrono::steady_clock::now();
#endif
    multidot_kernel<<<grid_size, block_size>>>(
        0, dim_size[0], dim_size[1],
        as_cuda_type(next_krylov_basis->get_const_values()), stride_next_krylov,
        as_cuda_accessor(next_krylov_const_accessor),
        as_cuda_type(arnoldi_norm->get_values() + dim_size[1]), 0,
        as_cuda_type(stop_status));
#ifdef TIMING
    exec->synchronize();
    auto time_3 = std::chrono::steady_clock::now() - start_3;
    std::cout << "time_3(" << iter << ") = "
              << std::chrono::duration_cast<double_seconds>(time_3).count()
              << std::endl;
#endif
    // nrmN = norm(next_krylov_basis)
    components::fill_array(exec, num_reorth->get_data(), 1, zero<size_type>());
#ifdef TIMING
    exec->synchronize();
    auto start_4 = std::chrono::steady_clock::now();
#endif
    check_arnoldi_norms<default_block_size>
        <<<ceildiv(dim_size[1], default_block_size), default_block_size>>>(
            as_cuda_type(arnoldi_norm->get_const_values()), stride_arnoldi,
            as_cuda_type(arnoldi_norm->get_const_values() + stride_arnoldi),
            stride_arnoldi, as_cuda_type(stop_status),
            as_cuda_type(reorth_status), as_cuda_type(num_reorth->get_data()));
#ifdef TIMING
    exec->synchronize();
    auto time_4 = std::chrono::steady_clock::now() - start_4;
    std::cout << "time_4(" << iter << ") = "
              << std::chrono::duration_cast<double_seconds>(time_4).count()
              << std::endl;
#endif
#ifdef TIMING
    exec->synchronize();
    auto start_5 = std::chrono::steady_clock::now();
#endif
    numReorth = 0;
    exec->get_master()->copy_from(exec.get(), 1, num_reorth->get_const_data(),
                                  &numReorth);
#ifdef TIMING
    exec->synchronize();
    auto time_5 = std::chrono::steady_clock::now() - start_5;
    std::cout << "time_5(" << iter << ") = "
              << std::chrono::duration_cast<double_seconds>(time_5).count()
              << std::endl;
#endif
    // numReorth <= number of next_krylov vector to be reorthogonalization
    for (size_type l = 1; (numReorth > 0) && (l < 3); l++) {
        (*num_reorth_steps)++;
        (*num_reorth_vectors) += iter;
        zero_matrix(iter + 1, dim_size[1], stride_buffer,
                    buffer_iter->get_values());
        /*
        multidot_kernel_num_iters<<<grid_size, block_size>>>(
            iter + 1, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_const_values()),
            stride_next_krylov, as_cuda_accessor(krylov_bases.to_const()),
            as_cuda_type(buffer_iter->get_values()), stride_buffer,
            as_cuda_type(stop_status));
        */
        multidot_kernel_num_iters_1<<<grid_size_num_iters, block_size>>>(
            iter + 1, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_const_values()),
            stride_next_krylov, as_cuda_accessor(krylov_bases.to_const()),
            as_cuda_type(buffer_iter->get_values()), stride_buffer,
            as_cuda_type(stop_status));
        // for i in 1:iter
        //     hessenberg(iter, i) = next_krylov_basis' * krylov_bases(:, i)
        // end
        /*
        update_next_krylov_kernel_num_iters_and_add<default_block_size>
            <<<ceildiv(dim_size[0] * stride_next_krylov, default_block_size),
               default_block_size>>>(
                iter + 1, dim_size[0], dim_size[1],
                as_cuda_type(next_krylov_basis->get_values()),
                stride_next_krylov, as_cuda_accessor(krylov_bases.to_const()),
                as_cuda_type(hessenberg_iter->get_values()), stride_hessenberg,
                as_cuda_type(buffer_iter->get_const_values()), stride_buffer,
                as_cuda_type(stop_status), as_cuda_type(reorth_status));
        */ /* */
        update_next_krylov_kernel_num_iters_and_add_1<default_update_dim>
            <<<dim_size[0] * stride_next_krylov, default_update_dim>>>(
                iter + 1, dim_size[0], dim_size[1],
                as_cuda_type(next_krylov_basis->get_values()),
                stride_next_krylov, as_cuda_accessor(krylov_bases.to_const()),
                as_cuda_type(hessenberg_iter->get_values()), stride_hessenberg,
                as_cuda_type(buffer_iter->get_const_values()), stride_buffer,
                as_cuda_type(stop_status), as_cuda_type(reorth_status));
        /* */
        // for i in 1:iter
        //     next_krylov_basis  -= hessenberg(iter, i) * krylov_bases(:, i)
        // end
        components::fill_array(
            exec, arnoldi_norm->get_values() + dim_size[1] * (l - 1),
            dim_size[1], zero<ValueType>());
        multidot_kernel<<<grid_size, block_size>>>(
            0, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_const_values()),
            stride_next_krylov, as_cuda_accessor(next_krylov_const_accessor),
            as_cuda_type(arnoldi_norm->get_values() + dim_size[1] * (l - 1)), 0,
            as_cuda_type(stop_status));
        // nrmN = norm(next_krylov_basis)
        components::fill_array(exec, num_reorth->get_data(), 1,
                               zero<size_type>());
        check_arnoldi_norms<default_block_size>
            <<<ceildiv(dim_size[1], default_block_size), default_block_size>>>(
                as_cuda_type(arnoldi_norm->get_const_values() +
                             stride_arnoldi * (2 - l)),
                stride_arnoldi,
                as_cuda_type(arnoldi_norm->get_const_values() +
                             stride_arnoldi * (l - 1)),
                stride_arnoldi, as_cuda_type(stop_status),
                as_cuda_type(reorth_status),
                as_cuda_type(num_reorth->get_data()));
        numReorth = 0;
        exec->get_master()->copy_from(exec.get(), 1,
                                      num_reorth->get_const_data(), &numReorth);
        // numReorth <= number of next_krylov vector to be reorthogonalization
    }

#ifdef TIMING
    exec->synchronize();
    auto start_6 = std::chrono::steady_clock::now();
#endif
    update_hessenberg_2_kernel<default_block_size>
        <<<dim_size[1], default_block_size>>>(
            iter, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_const_values()),
            stride_next_krylov, as_cuda_type(hessenberg_iter->get_values()),
            stride_hessenberg, as_cuda_type(stop_status));
    update_krylov_next_krylov_kernel<default_block_size>
        <<<ceildiv(dim_size[0] * stride_next_krylov, default_block_size),
           default_block_size>>>(
            iter, dim_size[0], dim_size[1],
            as_cuda_type(next_krylov_basis->get_values()), stride_next_krylov,
            as_cuda_accessor(krylov_bases),
            as_cuda_type(hessenberg_iter->get_const_values()),
            stride_hessenberg, as_cuda_type(stop_status));
#ifdef TIMING
    exec->synchronize();
    auto time_6 = std::chrono::steady_clock::now() - start_6;
    std::cout << "time_6(" << iter << ") = "
              << std::chrono::duration_cast<double_seconds>(time_6).count()
              << std::endl;
#endif
    // next_krylov_basis /= hessenberg(iter, iter + 1)
    // krylov_bases(:, iter + 1) = next_krylov_basis
    // End of arnoldi
}
/**/

template <typename ValueType>
void givens_rotation(std::shared_ptr<const CudaExecutor> exec,
                     matrix::Dense<ValueType> *givens_sin,
                     matrix::Dense<ValueType> *givens_cos,
                     matrix::Dense<ValueType> *hessenberg_iter,
                     matrix::Dense<ValueType> *residual_norm,
                     matrix::Dense<ValueType> *residual_norm_collection,
                     const matrix::Dense<ValueType> *b_norm, size_type iter,
                     const Array<stopping_status> *stop_status)
{
    // TODO: tune block_size for optimal performance
    constexpr auto block_size = default_block_size;
    const auto num_cols = hessenberg_iter->get_size()[1];
    const dim3 block_dim{block_size, 1, 1};
    const dim3 grid_dim{
        static_cast<unsigned int>(ceildiv(num_cols, block_size)), 1, 1};

    givens_rotation_kernel<block_size><<<grid_dim, block_dim>>>(
        hessenberg_iter->get_size()[0], hessenberg_iter->get_size()[1], iter,
        as_cuda_type(hessenberg_iter->get_values()),
        hessenberg_iter->get_stride(), as_cuda_type(givens_sin->get_values()),
        givens_sin->get_stride(), as_cuda_type(givens_cos->get_values()),
        givens_cos->get_stride(), as_cuda_type(residual_norm->get_values()),
        as_cuda_type(residual_norm_collection->get_values()),
        residual_norm_collection->get_stride(),
        as_cuda_type(b_norm->get_const_values()),
        as_cuda_type(stop_status->get_const_data()));
}


template <typename ValueType, typename ValueTypeKrylovBases>
void step_1(std::shared_ptr<const CudaExecutor> exec,
            matrix::Dense<ValueType> *next_krylov_basis,
            matrix::Dense<ValueType> *givens_sin,
            matrix::Dense<ValueType> *givens_cos,
            matrix::Dense<ValueType> *residual_norm,
            matrix::Dense<ValueType> *residual_norm_collection,
            Accessor2d<ValueTypeKrylovBases, ValueType> krylov_bases,
            matrix::Dense<ValueType> *hessenberg_iter,
            matrix::Dense<ValueType> *buffer_iter,
            const matrix::Dense<ValueType> *b_norm,
            matrix::Dense<ValueType> *arnoldi_norm, size_type iter,
            Array<size_type> *final_iter_nums,
            const Array<stopping_status> *stop_status,
            Array<stopping_status> *reorth_status, Array<size_type> *num_reorth,
            int *num_reorth_steps, int *num_reorth_vectors)
{
    increase_final_iteration_numbers_kernel<<<
        static_cast<unsigned int>(
            ceildiv(final_iter_nums->get_num_elems(), default_block_size)),
        default_block_size>>>(as_cuda_type(final_iter_nums->get_data()),
                              as_cuda_type(stop_status->get_const_data()),
                              final_iter_nums->get_num_elems());
    //    finish_arnoldi(exec, next_krylov_basis, krylov_bases, hessenberg_iter,
    //                   iter, stop_status->get_const_data());
    //    finish_arnoldi_reorth(exec, next_krylov_basis, krylov_bases,
    //                          hessenberg_iter, buffer_iter, arnoldi_norm,
    //                          iter, stop_status->get_const_data(),
    //                          reorth_status->get_data(), num_reorth);
    //    finish_arnoldi_CGS(exec, next_krylov_basis, krylov_bases,
    //    hessenberg_iter,
    //                       buffer_iter, arnoldi_norm, iter,
    //                       stop_status->get_const_data(),
    //                       reorth_status->get_data(), num_reorth);
    finish_arnoldi_CGS2(exec, next_krylov_basis, krylov_bases, hessenberg_iter,
                        buffer_iter, arnoldi_norm, iter,
                        stop_status->get_const_data(),
                        reorth_status->get_data(), num_reorth, num_reorth_steps,
                        num_reorth_vectors);
    givens_rotation(exec, givens_sin, givens_cos, hessenberg_iter,
                    residual_norm, residual_norm_collection, b_norm, iter,
                    stop_status);
}

GKO_INSTANTIATE_FOR_EACH_GMRES_MIXED_TYPE(
    GKO_DECLARE_GMRES_MIXED_STEP_1_KERNEL);


template <typename ValueType>
void solve_upper_triangular(
    const matrix::Dense<ValueType> *residual_norm_collection,
    const matrix::Dense<ValueType> *hessenberg, matrix::Dense<ValueType> *y,
    const Array<size_type> *final_iter_nums)
{
    // TODO: tune block_size for optimal performance
    constexpr auto block_size = default_block_size;
    const auto num_rhs = residual_norm_collection->get_size()[1];
    const dim3 block_dim{block_size, 1, 1};
    const dim3 grid_dim{static_cast<unsigned int>(ceildiv(num_rhs, block_size)),
                        1, 1};

    solve_upper_triangular_kernel<block_size><<<grid_dim, block_dim>>>(
        hessenberg->get_size()[1], num_rhs,
        as_cuda_type(residual_norm_collection->get_const_values()),
        residual_norm_collection->get_stride(),
        as_cuda_type(hessenberg->get_const_values()), hessenberg->get_stride(),
        as_cuda_type(y->get_values()), y->get_stride(),
        as_cuda_type(final_iter_nums->get_const_data()));
}


template <typename ValueType, typename ValueTypeKrylovBases>
void calculate_qy(Accessor2dConst<ValueTypeKrylovBases, ValueType> krylov_bases,
                  size_type krylov_cols, const matrix::Dense<ValueType> *y,
                  matrix::Dense<ValueType> *before_preconditioner,
                  const Array<size_type> *final_iter_nums)
{
    const auto num_rows = before_preconditioner->get_size()[0];
    const auto num_cols = krylov_cols;  // krylov_bases->get_size()[1];
    const auto num_rhs = before_preconditioner->get_size()[1];
    const auto stride_before_preconditioner =
        before_preconditioner->get_stride();

    constexpr auto block_size = default_block_size;
    const dim3 grid_dim{
        static_cast<unsigned int>(
            ceildiv(num_rows * stride_before_preconditioner, block_size)),
        1, 1};
    const dim3 block_dim{block_size, 1, 1};


    calculate_Qy_kernel<block_size><<<grid_dim, block_dim>>>(
        num_rows, num_cols, num_rhs, as_cuda_accessor(krylov_bases),
        as_cuda_type(y->get_const_values()), y->get_stride(),
        as_cuda_type(before_preconditioner->get_values()),
        stride_before_preconditioner,
        as_cuda_type(final_iter_nums->get_const_data()));
    // Calculate qy
    // before_preconditioner = krylov_bases * y
}


template <typename ValueType, typename ValueTypeKrylovBases>
void step_2(std::shared_ptr<const CudaExecutor> exec,
            const matrix::Dense<ValueType> *residual_norm_collection,
            Accessor2dConst<ValueTypeKrylovBases, ValueType> krylov_bases,
            const matrix::Dense<ValueType> *hessenberg,
            matrix::Dense<ValueType> *y,
            matrix::Dense<ValueType> *before_preconditioner,
            const Array<size_type> *final_iter_nums)
{
    const auto res_norm_col_size = residual_norm_collection->get_size();
    const auto krylov_cols = res_norm_col_size[0] * res_norm_col_size[1];
    solve_upper_triangular(residual_norm_collection, hessenberg, y,
                           final_iter_nums);
    calculate_qy(krylov_bases, krylov_cols, y, before_preconditioner,
                 final_iter_nums);
}

GKO_INSTANTIATE_FOR_EACH_GMRES_MIXED_TYPE(
    GKO_DECLARE_GMRES_MIXED_STEP_2_KERNEL);


}  // namespace gmres_mixed
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
