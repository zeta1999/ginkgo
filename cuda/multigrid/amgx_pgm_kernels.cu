/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2020, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/multigrid/amgx_pgm_kernels.hpp"


#include <memory>


#include <hip/hip_runtime.h>
#include <hipsparse.h>


#include <ginkgo/core/base/exception_helpers.hpp>
#include <ginkgo/core/base/math.hpp>
#include <ginkgo/core/multigrid/amgx_pgm.hpp>


#include "core/components/fill_array.hpp"
#include "core/components/prefix_sum.hpp"
#include "cuda/base/cusparse_bindings.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"
#include "cuda/components/atomic.cuh"
#include "cuda/components/reduction.cuh"
#include "cuda/components/thread_ids.cuh"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The AMGX_PGM solver namespace.
 *
 * @ingroup amgx_pgm
 */
namespace amgx_pgm {


constexpr int default_block_size = 512;


#include "common/multigrid/amgx_pgm_kernels.hpp.inc"


template <typename ValueType, typename IndexType>
void restrict_apply(std::shared_ptr<const CudaExecutor> exec,
                    const Array<IndexType> &agg,
                    const matrix::Dense<ValueType> *b,
                    matrix::Dense<ValueType> *x)
{
    const dim3 grid(
        ceildiv(b->get_size()[0] * b->get_size()[1], default_block_size));
    components::fill_array(exec, x->get_values(), x->get_num_stored_elements(),
                           zero<ValueType>());
    kernel::restrict_apply_kernel<<<grid, default_block_size>>>(
        agg.get_const_data(), b->get_size()[0], b->get_size()[1],
        as_cuda_type(b->get_const_values()), b->get_stride(),
        as_cuda_type(x->get_values()), x->get_stride());
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_AMGX_PGM_RESTRICT_APPLY_KERNEL);


template <typename ValueType, typename IndexType>
void prolong_applyadd(std::shared_ptr<const CudaExecutor> exec,
                      const Array<IndexType> &agg,
                      const matrix::Dense<ValueType> *b,
                      matrix::Dense<ValueType> *x)
{
    const dim3 grid(
        ceildiv(x->get_size()[0] * x->get_size()[1], default_block_size));
    kernel::prolong_applyadd_kernel<<<grid, default_block_size>>>(
        agg.get_const_data(), x->get_size()[0], x->get_size()[1],
        as_cuda_type(b->get_const_values()), b->get_stride(),
        as_cuda_type(x->get_values()), x->get_stride());
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_AMGX_PGM_PROLONGATE_APPLY_KERNEL);


template <typename IndexType>
void match_edge(std::shared_ptr<const CudaExecutor> exec,
                const Array<IndexType> &strongest_neighbor,
                Array<IndexType> &agg)
{
    const auto num = agg.get_num_elems();
    const dim3 grid(ceildiv(num, default_block_size));
    kernel::match_edge_kernel<<<grid, default_block_size>>>(
        num, strongest_neighbor.get_const_data(), agg.get_data());
}

GKO_INSTANTIATE_FOR_EACH_INDEX_TYPE(GKO_DECLARE_AMGX_PGM_MATCH_EDGE_KERNEL);


template <typename IndexType>
void count_unagg(std::shared_ptr<const CudaExecutor> exec,
                 const Array<IndexType> &agg, size_type *num_unagg)
{
    Array<IndexType> active_agg(exec, agg.get_num_elems());
    const dim3 grid(ceildiv(active_agg.get_num_elems(), default_block_size));
    kernel::activate_kernel<<<grid, default_block_size>>>(
        active_agg.get_num_elems(), agg.get_const_data(),
        active_agg.get_data());
    *num_unagg = reduce_add_array(exec, active_agg.get_num_elems(),
                                  active_agg.get_const_data());
}

GKO_INSTANTIATE_FOR_EACH_INDEX_TYPE(GKO_DECLARE_AMGX_PGM_COUNT_UNAGG_KERNEL);


template <typename IndexType>
void renumber(std::shared_ptr<const CudaExecutor> exec, Array<IndexType> &agg,
              size_type *num_agg)
{
    const auto num = agg.get_num_elems();
    Array<IndexType> agg_map(exec, num + 1);
    components::fill_array(exec, agg_map.get_data(), agg_map.get_num_elems(),
                           zero<IndexType>());
    const dim3 grid(ceildiv(num, default_block_size));
    kernel::fill_agg_kernel<<<grid, default_block_size>>>(
        num, agg.get_const_data(), agg_map.get_data());
    components::prefix_sum(exec, agg_map.get_data(), agg_map.get_num_elems());
    kernel::renumber_kernel<<<grid, default_block_size>>>(
        num, agg_map.get_const_data(), agg.get_data());
    *num_agg = exec->copy_val_to_host(agg_map.get_const_data() + num);
}

GKO_INSTANTIATE_FOR_EACH_INDEX_TYPE(GKO_DECLARE_AMGX_PGM_RENUMBER_KERNEL);


template <typename ValueType, typename IndexType>
void find_strongest_neighbor(
    std::shared_ptr<const CudaExecutor> exec,
    const matrix::Csr<ValueType, IndexType> *weight_mtx,
    const matrix::Dense<ValueType> *diag, Array<IndexType> &agg,
    Array<IndexType> &strongest_neighbor)
{
    const auto num = agg.get_num_elems();
    const dim3 grid(ceildiv(num, default_block_size));
    kernel::find_strongest_neighbor_kernel<<<grid, default_block_size>>>(
        num, weight_mtx->get_const_row_ptrs(), weight_mtx->get_const_col_idxs(),
        weight_mtx->get_const_values(), diag->get_const_values(),
        diag->get_stride(), agg.get_data(), strongest_neighbor.get_data());
}

GKO_INSTANTIATE_FOR_EACH_NON_COMPLEX_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_AMGX_PGM_FIND_STRONGEST_NEIGHBOR);


template <typename ValueType, typename IndexType>
void assign_to_exist_agg(std::shared_ptr<const CudaExecutor> exec,
                         const matrix::Csr<ValueType, IndexType> *weight_mtx,
                         const matrix::Dense<ValueType> *diag,
                         Array<IndexType> &agg,
                         Array<IndexType> &intermediate_agg)
{
    auto agg_val = (intermediate_agg.get_num_elems() > 0)
                       ? intermediate_agg.get_data()
                       : agg.get_data();
    const auto num = agg.get_num_elems();
    const dim3 grid(ceildiv(num, default_block_size));
    kernel::assign_to_exist_agg_kernel<<<grid, default_block_size>>>(
        num, weight_mtx->get_const_row_ptrs(), weight_mtx->get_const_col_idxs(),
        weight_mtx->get_const_values(), diag->get_const_values(),
        diag->get_stride(), agg.get_const_data(), agg_val);
    if (intermediate_agg.get_num_elems() > 0) {
        // Copy the intermediate_agg to agg
        agg = intermediate_agg;
    }
}

GKO_INSTANTIATE_FOR_EACH_NON_COMPLEX_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_AMGX_PGM_ASSIGN_TO_EXIST_AGG);


template <typename ValueType, typename IndexType>
void amgx_pgm_generate(std::shared_ptr<const CudaExecutor> exec,
                       const matrix::Csr<ValueType, IndexType> *source,
                       const Array<IndexType> &agg,
                       matrix::Csr<ValueType, IndexType> *coarse)
{
    GKO_NOT_IMPLEMENTED;
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(GKO_DECLARE_AMGX_PGM_GENERATE);


}  // namespace amgx_pgm
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
